#include "hip/hip_runtime.h"
#include <vector>
#include <memory>
#include <iostream>
#include "imagem.cuh"
#include "vec3.cuh"
#include "scene.cuh"
#include "camera.cuh"
#include "shape.cuh"
#include "renderer.cuh"
#include "utilities.cuh"
#include "ray.cuh"
#include "material.cuh"
#include "config_reader.cuh"

__device__ color ray_color(const ray& r, Shape** objects, int n_obj, float t_min, float t_max, int depth, hiprandState curand_St)
{
    if (depth <= 0)
        return color(0,0,0);

    ray cur_ray = r;
    color cur_attenuation = color(1.0,1.0,1.0);

    bool any_hit = false;
    float min_t = 0;
    float tmp;
    Shape* closest_hit = nullptr;
    Shape* shape;

    for(int j = depth; j > 0; j--){
        any_hit = false;
        min_t = 0;
        closest_hit = nullptr;

        for(int i = 0; i < n_obj; i++){
            shape = objects[i];
            tmp = shape->hit(cur_ray, t_min, t_max);
            
            if((tmp < min_t || !any_hit) && tmp >= 0){
                min_t = tmp;
                closest_hit = shape;
                any_hit = true;
            }
        }

        if(any_hit){
            point3 p = r.at(min_t);
            vec3 n = closest_hit->normal(cur_ray, p);
            point3 target = p + n + random_in_unit_sphere(&curand_St);

            ray scattered;
            color attenuation;
            if (closest_hit->obj_material->scatter(cur_ray, n, p, cur_attenuation, scattered, &curand_St)){
                cur_attenuation *= attenuation;
                cur_ray = scattered;
                //return attenuation * ray_color(scattered, objects, n_obj, t_min, t_max, depth-1, curand_St);
            }
                
            return color(0,0,0);
        }
        else{
            vec3 unit_direction = unit_vector(r.direction());
            auto t = 0.5f*(unit_direction.y() + 1.0f);
            color c = (1.0f-t)*color(1.0f, 1.0f, 1.0f) + t*color(0.5f, 0.7f, 1.0f);
            return cur_attenuation*c;
        }
    }

    return color(0.0,0.0,0.0);
}

__global__ void fill_colors(
    color *matrix, int height, int width, Camera* camera, Shape** objects, int n_objs, int samples_per_pixel, float t_min, float t_max, int depth, hiprandState *curand_St
    ){
    hiprandState curand_States = curand_St[threadIdx.x + blockIdx.x * blockDim.x];

    for (int j = threadIdx.y + blockIdx.y * blockDim.y; j < height; j++) {
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < width; ++i) {
            printf("%d, %d\n", j, i);
            curand_States = curand_St[j*height+i];
            auto u = float(i) / (width-1);
            auto v = float(j) / (height-1);
            color pixel_color = color();
            
            for(int k = 0; k < samples_per_pixel; k++){
                pixel_color += ray_color(camera->get_ray(u+hiprand_uniform(&curand_States)/(width-1), 
                                                         v+hiprand_uniform(&curand_States)/(height-1),
                                                         &curand_States),
                                         objects,
                                         n_objs,
                                         t_min,
                                         t_max,
                                         depth,
                                         curand_States);
            }
            
            pixel_color /= samples_per_pixel;
            matrix[j*height+i] = pixel_color;
        }
    }
    
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x+threadIdx.y + blockIdx.y * blockDim.y;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__host__ std::shared_ptr<Imagem> Renderer::render(
    Scene* scene, int samples_per_pixel, float t_min, float t_max, int depth, bool log
    ){
    auto height = scene->image_height;
    auto width = scene->image_width;
    auto camera = scene->camera;
    auto objects = scene->objects;
    int n_obj = scene->n_obj;

    dim3 blocks(width/TX+1,height/TY+1);
    dim3 threads(TX,TY);

    color* matrix;

    checkCudaErrors(hipMallocManaged((void **)&matrix, height*width*sizeof(color)));

    hiprandState *curand_States = nullptr;
    checkCudaErrors(hipMalloc((void **)&curand_States, height*width*sizeof(hiprandState)));

    setup_kernel<<<blocks, threads>>>(curand_States);

    hipError_t err = hipGetLastError();        // Get error code
    if ( err != hipSuccess )
    {
      std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
      exit(-1);
    }

    checkCudaErrors(hipDeviceSynchronize());

    fill_colors<<<blocks, threads>>>(matrix, height, width, camera, objects, n_obj, samples_per_pixel, t_min, t_max, depth, curand_States);

    err = hipGetLastError();
    if ( err != hipSuccess ){
        std::cout << "CUDA Error:" << hipGetErrorString(err) << std::endl;       
        exit(-1);
    }
    checkCudaErrors(hipDeviceSynchronize());
    
    std::shared_ptr<Imagem> image (new Imagem(matrix, height, width));
    checkCudaErrors(hipFree(curand_States));

    return image;
}

int main() {
    // Criando cena
    SceneParams params = read_scene_from("../examples/config.scene");
    Renderer renderer;

    std::shared_ptr<Imagem> generated_image = renderer.render(  params.scene, 
                                                                params.samples_per_pixel,
                                                                0.001, 
                                                                infinity, 
                                                                params.max_depth, 
                                                                params.log);

    generated_image->salvar_imagem(params.filename.c_str());
    
    return 0;
}