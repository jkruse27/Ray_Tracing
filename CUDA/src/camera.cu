#include "hip/hip_runtime.h"
#include "camera.cuh"

__host__ Camera::Camera( float vp_height, 
                float vp_width,
                float focal_l,
                point3 orig,
                point3 vert,
                point3 hor,
                point3 lll
                ){
    this->viewport_height = vp_height;
    this->viewport_width = vp_width;
    this->focal_length = focal_l;
    this->origin = orig;
    this->vertical = vert;
    this->horizontal = hor;
    this->lower_left_corner = lll;
}

__host__ Camera::Camera(point3 origin, 
               point3 lookat,
               vec3 vup,
               float vfov,
               float aspect_ratio,
               float aperture,
               float focus_dist)
    {
    auto theta = degrees_to_radians(vfov);
    auto h = tan(theta/2);
    this->viewport_height = 2.0f * h;
    this->viewport_width = aspect_ratio * viewport_height;
    this->aperture = aperture;
    this->focus_dist = focus_dist;

    this->w = unit_vector(origin - lookat);
    this->u = unit_vector(cross(vup, w));
    this->v = cross(w, u);

    this->origin = origin;
    this->horizontal = focus_dist * this->viewport_width * this->u;
    this->vertical = focus_dist * this->viewport_height * this->v;
    this->lower_left_corner = origin - horizontal/2 - vertical/2 - (focus_dist* this->w);
}

__device__ ray Camera::get_ray(float x1, float x2, hiprandState *curand_States) {
    auto rd = (this->aperture/2) * random_in_unit_disk(curand_States);
    vec3 offset = this->u * rd.x() + this->v * rd.y();
    return ray(
                origin + offset, 
                lower_left_corner + x1*horizontal + x2*vertical - origin - offset
               );
}